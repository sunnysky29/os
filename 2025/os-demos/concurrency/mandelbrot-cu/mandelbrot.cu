#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define W 12800
#define H 12800
#define IMG_FILE "mandelbrot.ppm"
#define BLOCK_SIZE 16

int pic[W][H];

void write_ppm(FILE *fp, int step);

__device__ double mandelbrot(double x, double y) {
    int n = 0;
    double a = 0, b = 0, c, d;
    while ((c = a * a) + (d = b * b) < 4 && n++ < 880) {
        b = 2 * a * b + y * 1024 / H * 8e-9 - 0.645411;
        a = c - d + x * 1024 / W * 8e-9 + 0.356888;
    }
    return n;
}

__global__ void mandelbrot_kernel(int *pic_d) {
    // Calculate pixel coordinates from thread and block indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if within image bounds
    if (x < W && y < H) {
        // Compute Mandelbrot value for this pixel
        int n = mandelbrot(x, y);
        // Store result in global memory
        pic_d[y * W + x] = n;
    }
}

int main(int argc, char *argv[]) {
    // Allocate device memory for the result
    int *pic_d;
    hipMalloc(&pic_d, W * H * sizeof(int));
    
    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((W + BLOCK_SIZE - 1) / BLOCK_SIZE, (H + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Launch kernel
    printf("Rendering Mandelbrot set with CUDA (%d x %d pixels)...\n", W, H);
    float elapsed = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    mandelbrot_kernel<<<gridDim, blockDim>>>(pic_d);
    hipEventRecord(stop);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    // Copy result back to host
    hipMemcpy(pic, pic_d, W * H * sizeof(int), hipMemcpyDeviceToHost);
    
    // Calculate elapsed time
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Render time: %.1f ms\n", elapsed);
    
    // High-resolution final image
    FILE *fp = fopen(IMG_FILE, "w");
    assert(fp);
    write_ppm(fp, 2);
    fclose(fp);
    
    // Clean up
    hipFree(pic_d);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}

void write_ppm(FILE *fp, int step) { 
    // Portable Pixel Map (PPM)

    int w = W / step, h = H / step;

    fprintf(fp, "P6\n%d %d 255\n", w, h);
    for (int j = 0; j < H; j += step) {
        for (int i = 0; i < W; i += step) {
            int n = pic[i][j];
            int r = 255 * pow((n - 80) / 800.0, 3);
            int g = 255 * pow((n - 80) / 800.0, 0.7);
            int b = 255 * pow((n - 80) / 800.0, 0.5);
            fprintf(fp, "%c%c%c", r, g, b);
        }
    }
}